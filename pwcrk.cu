#include "hip/hip_runtime.h"
"""
LEGAL NOTICE:
This tool is intended for educational use only.
The author is not responsible for any misuse of this tool.
"""
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_LEN 8
#define HASH_LEN_MD5 16
#define HASH_LEN_SHA1 20
#define HASH_LEN_SHA256 32
#define CHARSET "abcdefghijklmnopqrstuvwxyz0123456789"
#define CHARSET_LEN 36

__device__ __constant__ char d_charset[CHARSET_LEN];
__device__ __constant__ uint8_t d_target_hash[HASH_LEN_SHA256];
__device__ __constant__ int d_hash_len;
__device__ __constant__ int d_pass_len;
__device__ __constant__ int d_hash_mode;
__device__ char d_result[MAX_LEN + 1];
__device__ int d_found = 0;

enum HashMode {
    HASH_MD5 = 0,
    HASH_SHA1 = 1,
    HASH_SHA256 = 2
};

// --- ROTL and ROTR ---
__device__ uint32_t ROTL(uint32_t x, int n) {
    return (x << n) | (x >> (32 - n));
}
__device__ uint32_t ROTR(uint32_t x, int n) {
    return (x >> n) | (x << (32 - n));
}

// --- MD5 Implementation ---
__device__ uint32_t md5_rotl(uint32_t x, int n) {
    return (x << n) | (x >> (32 - n));
}

__device__ void md5(const char* msg, int len, uint8_t* digest) {
    uint32_t a0 = 0x67452301;
    uint32_t b0 = 0xefcdab89;
    uint32_t c0 = 0x98badcfe;
    uint32_t d0 = 0x10325476;

    const uint32_t k[] = {
        0xd76aa478,0xe8c7b756,0x242070db,0xc1bdceee,
        0xf57c0faf,0x4787c62a,0xa8304613,0xfd469501,
        0x698098d8,0x8b44f7af,0xffff5bb1,0x895cd7be,
        0x6b901122,0xfd987193,0xa679438e,0x49b40821,
        0xf61e2562,0xc040b340,0x265e5a51,0xe9b6c7aa,
        0xd62f105d,0x02441453,0xd8a1e681,0xe7d3fbc8,
        0x21e1cde6,0xc33707d6,0xf4d50d87,0x455a14ed,
        0xa9e3e905,0xfcefa3f8,0x676f02d9,0x8d2a4c8a,
        0xfffa3942,0x8771f681,0x6d9d6122,0xfde5380c,
        0xa4beea44,0x4bdecfa9,0xf6bb4b60,0xbebfbc70,
        0x289b7ec6,0xeaa127fa,0xd4ef3085,0x04881d05,
        0xd9d4d039,0xe6db99e5,0x1fa27cf8,0xc4ac5665,
        0xf4292244,0x432aff97,0xab9423a7,0xfc93a039,
        0x655b59c3,0x8f0ccc92,0xffeff47d,0x85845dd1,
        0x6fa87e4f,0xfe2ce6e0,0xa3014314,0x4e0811a1,
        0xf7537e82,0xbd3af235,0x2ad7d2bb,0xeb86d391
    };

    const int r[] = {
        7,12,17,22,7,12,17,22,7,12,17,22,7,12,17,22,
        5,9,14,20,5,9,14,20,5,9,14,20,5,9,14,20,
        4,11,16,23,4,11,16,23,4,11,16,23,4,11,16,23,
        6,10,15,21,6,10,15,21,6,10,15,21,6,10,15,21
    };

    uint8_t data[64] = {0};
    for (int i = 0; i < len; ++i) data[i] = msg[i];
    data[len] = 0x80;
    uint64_t bit_len = len * 8;
    data[56] = bit_len & 0xFF;
    data[57] = (bit_len >> 8) & 0xFF;
    data[58] = (bit_len >> 16) & 0xFF;
    data[59] = (bit_len >> 24) & 0xFF;
    data[60] = (bit_len >> 32) & 0xFF;
    data[61] = (bit_len >> 40) & 0xFF;
    data[62] = (bit_len >> 48) & 0xFF;
    data[63] = (bit_len >> 56) & 0xFF;

    uint32_t M[16];
    for (int i = 0; i < 16; ++i)
        M[i] = (data[i*4]) | (data[i*4+1] << 8) | (data[i*4+2] << 16) | (data[i*4+3] << 24);

    uint32_t A = a0;
    uint32_t B = b0;
    uint32_t C = c0;
    uint32_t D = d0;

    for (int i = 0; i < 64; ++i) {
        uint32_t F, g;
        if (i < 16) {
            F = (B & C) | ((~B) & D);
            g = i;
        } else if (i < 32) {
            F = (D & B) | ((~D) & C);
            g = (5 * i + 1) & 15;
        } else if (i < 48) {
            F = B ^ C ^ D;
            g = (3 * i + 5) & 15;
        } else {
            F = C ^ (B | (~D));
            g = (7 * i) & 15;
        }
        uint32_t temp = D;
        D = C;
        C = B;
        B = B + md5_rotl(A + F + k[i] + M[g], r[i]);
        A = temp;
    }

    a0 += A;
    b0 += B;
    c0 += C;
    d0 += D;

    digest[0] = a0 & 0xff; digest[1] = (a0 >> 8) & 0xff; digest[2] = (a0 >> 16) & 0xff; digest[3] = (a0 >> 24) & 0xff;
    digest[4] = b0 & 0xff; digest[5] = (b0 >> 8) & 0xff; digest[6] = (b0 >> 16) & 0xff; digest[7] = (b0 >> 24) & 0xff;
    digest[8] = c0 & 0xff; digest[9] = (c0 >> 8) & 0xff; digest[10] = (c0 >> 16) & 0xff; digest[11] = (c0 >> 24) & 0xff;
    digest[12] = d0 & 0xff; digest[13] = (d0 >> 8) & 0xff; digest[14] = (d0 >> 16) & 0xff; digest[15] = (d0 >> 24) & 0xff;
}

// SHA1 device implementation
__device__ uint32_t ROTL32(uint32_t x, uint32_t n) {
    return (x << n) | (x >> (32 - n));
}

__device__ void sha1(const char* msg, int len, uint8_t* digest) {
    uint32_t h0 = 0x67452301, h1 = 0xEFCDAB89, h2 = 0x98BADCFE, h3 = 0x10325476, h4 = 0xC3D2E1F0;
    uint8_t data[64] = {0};

    for (int i = 0; i < len; ++i) data[i] = msg[i];
    data[len] = 0x80;
    data[63] = len * 8;

    uint32_t w[80];
    for (int i = 0; i < 16; ++i)
        w[i] = (data[4*i]<<24) | (data[4*i+1]<<16) | (data[4*i+2]<<8) | (data[4*i+3]);
    for (int i = 16; i < 80; ++i)
        w[i] = ROTL32(w[i-3] ^ w[i-8] ^ w[i-14] ^ w[i-16], 1);

    uint32_t a = h0, b = h1, c = h2, d = h3, e = h4;

    for (int i = 0; i < 80; ++i) {
        uint32_t f, k;
        if (i < 20) {
            f = (b & c) | ((~b) & d);
            k = 0x5A827999;
        } else if (i < 40) {
            f = b ^ c ^ d;
            k = 0x6ED9EBA1;
        } else if (i < 60) {
            f = (b & c) | (b & d) | (c & d);
            k = 0x8F1BBCDC;
        } else {
            f = b ^ c ^ d;
            k = 0xCA62C1D6;
        }
        uint32_t temp = ROTL32(a,5) + f + e + k + w[i];
        e = d;
        d = c;
        c = ROTL32(b,30);
        b = a;
        a = temp;
    }

    h0 += a; h1 += b; h2 += c; h3 += d; h4 += e;

    digest[0] = (h0 >> 24) & 0xFF;
    digest[1] = (h0 >> 16) & 0xFF;
    digest[2] = (h0 >> 8) & 0xFF;
    digest[3] = h0 & 0xFF;

    digest[4] = (h1 >> 24) & 0xFF;
    digest[5] = (h1 >> 16) & 0xFF;
    digest[6] = (h1 >> 8) & 0xFF;
    digest[7] = h1 & 0xFF;

    digest[8] = (h2 >> 24) & 0xFF;
    digest[9] = (h2 >> 16) & 0xFF;
    digest[10] = (h2 >> 8) & 0xFF;
    digest[11] = h2 & 0xFF;

    digest[12] = (h3 >> 24) & 0xFF;
    digest[13] = (h3 >> 16) & 0xFF;
    digest[14] = (h3 >> 8) & 0xFF;
    digest[15] = h3 & 0xFF;

    digest[16] = (h4 >> 24) & 0xFF;
    digest[17] = (h4 >> 16) & 0xFF;
    digest[18] = (h4 >> 8) & 0xFF;
    digest[19] = h4 & 0xFF;
}

// SHA256 device implementation
__device__ uint32_t SIG0(uint32_t x) {
    return ROTR(x, 7) ^ ROTR(x, 18) ^ (x >> 3);
}
__device__ uint32_t SIG1(uint32_t x) {
    return ROTR(x, 17) ^ ROTR(x, 19) ^ (x >> 10);
}
__device__ uint32_t EP0(uint32_t x) {
    return ROTR(x, 2) ^ ROTR(x, 13) ^ ROTR(x, 22);
}
__device__ uint32_t EP1(uint32_t x) {
    return ROTR(x, 6) ^ ROTR(x, 11) ^ ROTR(x, 25);
}
__device__ void sha256(const char* msg, int len, uint8_t* digest) {
    const uint32_t k[64] = {
        0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,
        0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
        0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,
        0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
        0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,
        0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
        0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,
        0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
        0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,
        0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
        0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,
        0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
        0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,
        0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
        0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,
        0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
    };

    uint32_t h[8] = {
        0x6a09e667,0xbb67ae85,0x3c6ef372,0xa54ff53a,
        0x510e527f,0x9b05688c,0x1f83d9ab,0x5be0cd19
    };

    uint8_t data[64] = {0};
    for (int i = 0; i < len; ++i) data[i] = msg[i];
    data[len] = 0x80;
    uint64_t bit_len = len * 8;
    data[63] = bit_len & 0xFF;
    data[62] = (bit_len >> 8) & 0xFF;

    uint32_t w[64];
    for (int i = 0; i < 16; ++i)
        w[i] = (data[4*i]<<24) | (data[4*i+1]<<16) | (data[4*i+2]<<8) | (data[4*i+3]);
    for (int i = 16; i < 64; ++i)
        w[i] = SIG1(w[i-2]) + w[i-7] + SIG0(w[i-15]) + w[i-16];

    uint32_t a = h[0], b = h[1], c = h[2], d = h[3];
    uint32_t e = h[4], f = h[5], g = h[6], h_ = h[7];

    for (int i = 0; i < 64; ++i) {
        uint32_t t1 = h_ + EP1(e) + ((e & f) ^ (~e & g)) + k[i] + w[i];
        uint32_t t2 = EP0(a) + ((a & b) ^ (a & c) ^ (b & c));
        h_ = g; g = f; f = e; e = d + t1;
        d = c; c = b; b = a; a = t1 + t2;
    }

    h[0] += a; h[1] += b; h[2] += c; h[3] += d;
    h[4] += e; h[5] += f; h[6] += g; h[7] += h_;

    for (int i = 0; i < 8; ++i) {
        digest[i*4 + 0] = (h[i] >> 24) & 0xFF;
        digest[i*4 + 1] = (h[i] >> 16) & 0xFF;
        digest[i*4 + 2] = (h[i] >> 8) & 0xFF;
        digest[i*4 + 3] = h[i] & 0xFF;
    }
}

// indexToPassword
__device__ void indexToPassword(uint64_t index, char* output, int length) {
    for (int i = length - 1; i >= 0; --i) {
        output[i] = d_charset[index % CHARSET_LEN];
        index /= CHARSET_LEN;
    }
}

// kernel
__global__ void bruteForceKernel(uint64_t offset, uint64_t total) {
    uint64_t idx = offset + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total || d_found) return;

    char pwd[MAX_LEN + 1] = {0};
    uint8_t digest[HASH_LEN_SHA256] = {0};
    indexToPassword(idx, pwd, d_pass_len);

    switch (d_hash_mode) {
        case HASH_MD5:
            md5(pwd, d_pass_len, digest);
            break;
        case HASH_SHA1:
            sha1(pwd, d_pass_len, digest);
            break;
        case HASH_SHA256:
            sha256(pwd, d_pass_len, digest);
            break;
    }

    bool match = true;
    for (int i = 0; i < d_hash_len; ++i) {
        if (digest[i] != d_target_hash[i]) {
            match = false;
            break;
        }
    }

    if (match && atomicExch(&d_found, 1) == 0) {
        for (int i = 0; i < d_pass_len; ++i)
            d_result[i] = pwd[i];
        d_result[d_pass_len] = '\0';
    }
}

// hex to bytes helper
void hexToBytes(const char* hex, uint8_t* out, int len) {
    for (int i = 0; i < len; ++i)
        sscanf(&hex[i * 2], "%2hhx", &out[i]);
}

const char* getHashName(int mode) {
    switch (mode) {
        case HASH_MD5: return "MD5";
        case HASH_SHA1: return "SHA1";
        case HASH_SHA256: return "SHA256";
        default: return "UNKNOWN";
    }
}

// main
int main(int argc, char** argv) {
    if (argc != 4) {
        printf("Usage: %s <hash> <length> <md5|sha1|sha256>\\n", argv[0]);
        return 1;
    }

    const char* hashHex = argv[1];
    int passLen = atoi(argv[2]);
    int hashLen;
    int mode;

    if (strcmp(argv[3], "md5") == 0) {
        mode = HASH_MD5;
        hashLen = HASH_LEN_MD5;
    } else if (strcmp(argv[3], "sha1") == 0) {
        mode = HASH_SHA1;
        hashLen = HASH_LEN_SHA1;
    } else if (strcmp(argv[3], "sha256") == 0) {
        mode = HASH_SHA256;
        hashLen = HASH_LEN_SHA256;
    } else {
        printf("Unsupported hash mode: %s\\n", argv[3]);
        return 1;
    }

    uint8_t hashBin[HASH_LEN_SHA256] = {0};
    hexToBytes(hashHex, hashBin, hashLen);

    hipMemcpyToSymbol(HIP_SYMBOL(d_target_hash), hashBin, hashLen, 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_charset), CHARSET, CHARSET_LEN, 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_hash_len), &hashLen, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_pass_len), &passLen, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_hash_mode), &mode, sizeof(int), 0, hipMemcpyHostToDevice);

    int zero = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(d_found), &zero, sizeof(int), 0, hipMemcpyHostToDevice);

    uint64_t total = 1;
    for (int i = 0; i < passLen; ++i) total *= CHARSET_LEN;

    printf("Starting %s brute-force (%llu keys)...\n", getHashName(mode), total);
    fflush(stdout);

    int threads = 256;
    int blocks = 256;

    for (uint64_t i = 0; i < total; i += threads * blocks) {
        bruteForceKernel<<<blocks, threads>>>(i, total);
        hipDeviceSynchronize();

        int found;
        hipMemcpyFromSymbol(&found, HIP_SYMBOL(d_found), sizeof(int));
        if (found) break;
    }

    char result[MAX_LEN + 1] = {0};
    hipMemcpyFromSymbol(result, HIP_SYMBOL(d_result), MAX_LEN + 1);

    if (result[0])
        printf("Password found: %s\n", result);
    else
        printf("Password not found in keyspace.\n");

    return 0;
}
